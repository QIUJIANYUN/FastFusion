#include "hip/hip_runtime.h"
// Copyright 2014-2017 Oxford University Innovation Limited and the authors of InfiniTAM

#include "ITMViewBuilder_CUDA.h"

#include "../Shared/ITMViewBuilder_Shared.h"
#include "../../../../ORUtils/CUDADefines.h"
#include "../../../../ORUtils/MemoryBlock.h"

using namespace ITMLib;
using namespace ORUtils;

ITMViewBuilder_CUDA::ITMViewBuilder_CUDA(const ITMRGBDCalib& calib):ITMViewBuilder(calib) { }
ITMViewBuilder_CUDA::~ITMViewBuilder_CUDA(void) { }

//---------------------------------------------------------------------------
//
// kernel function declaration 
//
//---------------------------------------------------------------------------

__global__ void alignDepth2Color_device(float *d_out, const float *d_in, const Matrix4f d2c, const Vector4f depth,
                                        const Vector4f color, Vector2i imgSize);
__global__ void convertDisparityToDepth_device(float *depth_out, const short *depth_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize);
__global__ void convertDepthAffineToFloat_device(float *d_out, const short *d_in, Vector2i imgSize, Vector2f depthCalibParams);
__global__ void filterDepth_device(float *imageData_out, const float *imageData_in, Vector2i imgDims);
__global__ void zr300_depth_denoise_piece_device(float *imageData_out, const float *imageData_in, Vector2i imgDims);
__global__ void zr300_depth_denoise_point_device(float *imageData_out, const float *imageData_in, Vector2i imgDims);
__global__ void filterBilateral_device(float *imageData_out, const float *imageData_in, Vector2i imgDims);
__global__ void ComputeNormalAndWeight_device(const float* depth_in, Vector4f* normal_out, float *sigmaL_out, Vector2i imgDims, Vector4f intrinsic);

//---------------------------------------------------------------------------
//
// host methods
//
//---------------------------------------------------------------------------

void ITMViewBuilder_CUDA::UpdateView(ITMView **view_ptr, ITMUChar4Image *rgbImage, ITMShortImage *rawDepthImage, bool useBilateralFilter, bool modelSensorNoise, bool storePreviousImage)
{
	if (*view_ptr == NULL)
	{
		*view_ptr = new ITMView(calib, rgbImage->noDims, rawDepthImage->noDims, true);
		if (this->shortImage != NULL) delete this->shortImage;
		this->shortImage = new ITMShortImage(rawDepthImage->noDims, true, true);
		if (this->floatImage != NULL) delete this->floatImage;
		this->floatImage = new ITMFloatImage(rawDepthImage->noDims, true, true);

		if (modelSensorNoise)
		{
			(*view_ptr)->depthNormal = new ITMFloat4Image(rawDepthImage->noDims, true, true);
			(*view_ptr)->depthUncertainty = new ITMFloatImage(rawDepthImage->noDims, true, true);
		}
	}

	ITMView *view = *view_ptr;

	if (storePreviousImage)
	{
		if (!view->rgb_prev) view->rgb_prev = new ITMUChar4Image(rgbImage->noDims, true, true);
		else view->rgb_prev->SetFrom(view->rgb, MemoryBlock<Vector4u>::CUDA_TO_CUDA);
	}	

	view->rgb->SetFrom(rgbImage, MemoryBlock<Vector4u>::CPU_TO_CUDA);
	this->shortImage->SetFrom(rawDepthImage, MemoryBlock<short>::CPU_TO_CUDA);

	switch (view->calib.disparityCalib.GetType())
	{
	case ITMDisparityCalib::TRAFO_KINECT:
		this->ConvertDisparityToDepth(view->depth, this->shortImage, &(view->calib.intrinsics_d), view->calib.disparityCalib.GetParams());
		break;
	case ITMDisparityCalib::TRAFO_AFFINE:
		this->ConvertDepthAffineToFloat(view->depth, this->shortImage, view->calib.disparityCalib.GetParams());
		break;
	default:
		break;
	}

	if (useBilateralFilter)
	{
		//3 step filtering
		//1. 双边滤波
		this->DepthFiltering( this->floatImage,view->depth);
		//2. 通过距离内有效点的数量滤波
		this->zr300_depth_denoise_point(view->depth, this->floatImage);
		//3. 去除噪点（片）
		this->zr300_depth_denoise_piece(this->floatImage, view->depth);
		view->depth->SetFrom(this->floatImage, MemoryBlock<float>::CUDA_TO_CUDA);
		view->depth->SetFrom(this->floatImage, MemoryBlock<float>::CUDA_TO_CPU);
	}

	if (modelSensorNoise)
	{
		this->ComputeNormalAndWeights(view->depthNormal, view->depthUncertainty, view->depth, view->calib.intrinsics_d.projectionParamsSimple.all);
	}
}

void ITMViewBuilder_CUDA::UpdateView(ITMView **view_ptr, ITMUChar4Image *rgbImage, ITMShortImage *depthImage, bool useBilateralFilter, ITMIMUMeasurement *imuMeasurement, bool modelSensorNoise, bool storePreviousImage)
{
	if (*view_ptr == NULL) 
	{
		*view_ptr = new ITMViewIMU(calib, rgbImage->noDims, depthImage->noDims, true);
		if (this->shortImage != NULL) delete this->shortImage;
		this->shortImage = new ITMShortImage(depthImage->noDims, true, true);
		if (this->floatImage != NULL) delete this->floatImage;
		this->floatImage = new ITMFloatImage(depthImage->noDims, true, true);

		if (modelSensorNoise)
		{
			(*view_ptr)->depthNormal = new ITMFloat4Image(depthImage->noDims, true, true);
			(*view_ptr)->depthUncertainty = new ITMFloatImage(depthImage->noDims, true, true);
		}
	}

	ITMViewIMU* imuView = (ITMViewIMU*)(*view_ptr);
	imuView->imu->SetFrom(imuMeasurement);

	this->UpdateView(view_ptr, rgbImage, depthImage, useBilateralFilter, modelSensorNoise, storePreviousImage);
}

void ITMViewBuilder_CUDA::UpdateView(ITMView **view_ptr, ITMUChar4Image *rgbImage, ITMShortImage *rawDepthImage, bool useBilateralFilter, cv::Mat *grayimg, std::vector<DataReader::IMUData> *relatedIMU, double imgtime, bool modelSensorNoise, bool storePreviousImage)
{
	if (*view_ptr == NULL)
	{
		*view_ptr = new ITMView(calib, rgbImage->noDims, rawDepthImage->noDims, true);
		if (this->shortImage != NULL) delete this->shortImage;
		this->shortImage = new ITMShortImage(rawDepthImage->noDims, true, true);
		if (this->floatImage != NULL) delete this->floatImage;
		this->floatImage = new ITMFloatImage(rawDepthImage->noDims, true, true);

		if (modelSensorNoise)
		{
			(*view_ptr)->depthNormal = new ITMFloat4Image(rawDepthImage->noDims, true, true);
			(*view_ptr)->depthUncertainty = new ITMFloatImage(rawDepthImage->noDims, true, true);
		}
	}

	ITMView *view = *view_ptr;

	view->grayimg = grayimg;
	view->imgtime = imgtime;
	view->relatedIMU = relatedIMU;

	if (storePreviousImage)
	{
		if (!view->rgb_prev) view->rgb_prev = new ITMUChar4Image(rgbImage->noDims, true, true);
		else view->rgb_prev->SetFrom(view->rgb, MemoryBlock<Vector4u>::CUDA_TO_CUDA);
	}

	view->rgb->SetFrom(rgbImage, MemoryBlock<Vector4u>::CPU_TO_CUDA);
	this->shortImage->SetFrom(rawDepthImage, MemoryBlock<short>::CPU_TO_CUDA);

	switch (view->calib.disparityCalib.GetType())
	{
		case ITMDisparityCalib::TRAFO_KINECT:
			this->ConvertDisparityToDepth(view->depth, this->shortImage, &(view->calib.intrinsics_d), view->calib.disparityCalib.GetParams());
			break;
		case ITMDisparityCalib::TRAFO_AFFINE:
			this->ConvertDepthAffineToFloat(view->depth, this->shortImage, view->calib.disparityCalib.GetParams());
			break;
		default:
			break;
	}

	if (useBilateralFilter)
	{
		//3 step filtering
		//1. 双边滤波
		this->DepthFiltering( this->floatImage,view->depth);
		//2. 通过距离内有效点的数量滤波
		this->zr300_depth_denoise_point(view->depth, this->floatImage);
		//3. 去除噪点（片）
		this->zr300_depth_denoise_piece(this->floatImage, view->depth);
		view->depth->SetFrom(this->floatImage, MemoryBlock<float>::CUDA_TO_CUDA);
		view->depth->SetFrom(this->floatImage, MemoryBlock<float>::CUDA_TO_CPU);
	}

    if (view->aligned_depth != NULL) delete view->aligned_depth;
    view->aligned_depth = new ITMFloatImage(rawDepthImage->noDims, true, true);
	this->depth_Align2_color(view->aligned_depth, view->depth, &(view->calib.trafo_depth_to_rgb), &(view->calib.intrinsics_d), &(view->calib.intrinsics_rgb));
	view->aligned_depth->SetFrom(view->aligned_depth, MemoryBlock<float>::CUDA_TO_CPU);

	if (modelSensorNoise)
	{
		this->ComputeNormalAndWeights(view->depthNormal, view->depthUncertainty, view->depth, view->calib.intrinsics_d.projectionParamsSimple.all);
	}
}

void ITMViewBuilder_CUDA::depth_Align2_color(ITMFloatImage *depth_out, const ITMFloatImage *depth_in, const ITMExtrinsics *depthExtrinsics, const ITMIntrinsics *depthIntrinsics, const ITMIntrinsics *colorIntrinsics)
{
	Vector2i imgSize = depth_in->noDims;

	const float *d_in = depth_in->GetData(MEMORYDEVICE_CUDA);
	float *d_out = depth_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	alignDepth2Color_device<< <gridSize, blockSize >> >(d_out, d_in, depthExtrinsics->calib, depthIntrinsics->projectionParamsSimple.all, colorIntrinsics->projectionParamsSimple.all, imgSize);
	ORcudaKernelCheck;
}

void ITMViewBuilder_CUDA::ConvertDisparityToDepth(ITMFloatImage *depth_out, const ITMShortImage *depth_in, const ITMIntrinsics *depthIntrinsics,
	Vector2f disparityCalibParams)
{
	Vector2i imgSize = depth_in->noDims;

	const short *d_in = depth_in->GetData(MEMORYDEVICE_CUDA);
	float *d_out = depth_out->GetData(MEMORYDEVICE_CUDA);

	float fx_depth = depthIntrinsics->projectionParamsSimple.fx;

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	convertDisparityToDepth_device << <gridSize, blockSize >> >(d_out, d_in, disparityCalibParams, fx_depth, imgSize);
	ORcudaKernelCheck;
}

void ITMViewBuilder_CUDA::ConvertDepthAffineToFloat(ITMFloatImage *depth_out, const ITMShortImage *depth_in, Vector2f depthCalibParams)
{
	Vector2i imgSize = depth_in->noDims;

	const short *d_in = depth_in->GetData(MEMORYDEVICE_CUDA);
	float *d_out = depth_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgSize.x / (float)blockSize.x), (int)ceil((float)imgSize.y / (float)blockSize.y));

	convertDepthAffineToFloat_device << <gridSize, blockSize >> >(d_out, d_in, imgSize, depthCalibParams);
	ORcudaKernelCheck;
}

void ITMViewBuilder_CUDA::DepthFiltering(ITMFloatImage *image_out, const ITMFloatImage *image_in)
{
	Vector2i imgDims = image_in->noDims;

	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgDims.x / (float)blockSize.x), (int)ceil((float)imgDims.y / (float)blockSize.y));

	filterDepth_device << <gridSize, blockSize >> >(imageData_out, imageData_in, imgDims);
	ORcudaKernelCheck;
}

void ITMViewBuilder_CUDA::zr300_depth_denoise_piece(ITMFloatImage *image_out, const ITMFloatImage *image_in)
{
	Vector2i imgDims = image_in->noDims;

	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgDims.x / (float)blockSize.x), (int)ceil((float)imgDims.y / (float)blockSize.y));

	zr300_depth_denoise_piece_device << <gridSize, blockSize >> >(imageData_out, imageData_in, imgDims);
}

void ITMViewBuilder_CUDA::zr300_depth_denoise_point(ITMFloatImage *image_out, const ITMFloatImage *image_in)
{
	Vector2i imgDims = image_in->noDims;

	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgDims.x / (float)blockSize.x), (int)ceil((float)imgDims.y / (float)blockSize.y));

	zr300_depth_denoise_point_device << <gridSize, blockSize >> >(imageData_out, imageData_in, imgDims);
}

void ITMViewBuilder_CUDA::BilateralFiltering(ITMFloatImage *image_out, const ITMFloatImage *image_in)
{
	Vector2i imgDims = image_in->noDims;

	const float *imageData_in = image_in->GetData(MEMORYDEVICE_CUDA);
	float *imageData_out = image_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgDims.x / (float)blockSize.x), (int)ceil((float)imgDims.y / (float)blockSize.y));

	filterBilateral_device << <gridSize, blockSize >> >(imageData_out, imageData_in, imgDims);
}

void ITMViewBuilder_CUDA::ComputeNormalAndWeights(ITMFloat4Image *normal_out, ITMFloatImage *sigmaZ_out, const ITMFloatImage *depth_in, Vector4f intrinsic)
{
	Vector2i imgDims = depth_in->noDims;

	const float *depthData_in = depth_in->GetData(MEMORYDEVICE_CUDA);

	float *sigmaZData_out = sigmaZ_out->GetData(MEMORYDEVICE_CUDA);
	Vector4f *normalData_out = normal_out->GetData(MEMORYDEVICE_CUDA);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)imgDims.x / (float)blockSize.x), (int)ceil((float)imgDims.y / (float)blockSize.y));

	ComputeNormalAndWeight_device << <gridSize, blockSize >> >(depthData_in, normalData_out, sigmaZData_out, imgDims, intrinsic);
	ORcudaKernelCheck;
}

//---------------------------------------------------------------------------
//
// kernel function implementation
//
//---------------------------------------------------------------------------

__global__ void alignDepth2Color_device(float *d_out, const float *d_in, const Matrix4f d2c, const Vector4f depth,
                                        const Vector4f color, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	int locId = x + y * imgSize.x;
	float d = d_in[locId];
    if(d<1e-3) return;

	float d1,d2,d3;
    d1 = (x - depth.z)*d/depth.x;
    d2 = (y - depth.w)*d/depth.y;
    d3 = d;

	float c1,c2,c3;
    c1 = d2c.m00*d1 + d2c.m10*d2 + d2c.m20*d3 + d2c.m30;
    c2 = d2c.m01*d1 + d2c.m11*d2 + d2c.m21*d3 + d2c.m31;
    c3 = d2c.m02*d1 + d2c.m12*d2 + d2c.m22*d3 + d2c.m32;

	int u = (int)(color.x * c1/c3 + color.z);
	int v = (int)(color.y * c2/c3 + color.w);

	if(u>=0 && u<imgSize.x && v>=0 && v<imgSize.y)
		d_out[u + v * imgSize.x] = d;
}
__global__ void convertDisparityToDepth_device(float *d_out, const short *d_in, Vector2f disparityCalibParams, float fx_depth, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	convertDisparityToDepth(d_out, x, y, d_in, disparityCalibParams, fx_depth, imgSize);
}

__global__ void convertDepthAffineToFloat_device(float *d_out, const short *d_in, Vector2i imgSize, Vector2f depthCalibParams)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if ((x >= imgSize.x) || (y >= imgSize.y)) return;

	convertDepthAffineToFloat(d_out, x, y, d_in, imgSize, depthCalibParams);
}

__global__ void filterDepth_device(float *imageData_out, const float *imageData_in, Vector2i imgDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 6 || x > imgDims.x - 6 || y < 6 || y > imgDims.y - 6) return;

	filterDepth(imageData_out, imageData_in, x, y, imgDims);
}

__global__ void zr300_depth_denoise_piece_device(float *imageData_out, const float *imageData_in, Vector2i imgDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 2 || x > imgDims.x - 2 || y < 2 || y > imgDims.y - 2) return;

	depth_denoise_piece(imageData_out, imageData_in, x, y, imgDims);
}

__global__ void zr300_depth_denoise_point_device(float *imageData_out, const float *imageData_in, Vector2i imgDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 3 || x > imgDims.x - 3 || y < 3 || y > imgDims.y - 3) return;

	depth_denoise_point(imageData_out, imageData_in, x, y, imgDims);
}

__global__ void filterBilateral_device(float *imageData_out, const float *imageData_in, Vector2i imgDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < 10 || x > imgDims.x - 10 || y < 10 || y > imgDims.y - 10) return;

	filterBilateral(imageData_out, imageData_in, x, y, imgDims);
}

__global__ void ComputeNormalAndWeight_device(const float* depth_in, Vector4f* normal_out, float *sigmaZ_out, Vector2i imgDims, Vector4f intrinsic)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	int idx = x + y * imgDims.x;

	if (x < 2 || x > imgDims.x - 2 || y < 2 || y > imgDims.y - 2)
	{
		normal_out[idx].w = -1.0f;
		sigmaZ_out[idx] = -1;
		return;
	}
	else
	{
		computeNormalAndWeight(depth_in, normal_out, sigmaZ_out, x, y, imgDims, intrinsic);
	}
}

